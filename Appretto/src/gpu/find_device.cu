#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

//this is copied from the CUDA sdk 
extern "C" int find_devices()
{
  int deviceCount;

  hipGetDeviceCount(&deviceCount);

  if(deviceCount==0) printf("There is no device supporting CUDA\n");
  for(int dev=0;dev<deviceCount;dev++)
    {
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp,dev);
      if(dev==0)
        {
          if(deviceProp.major==9999 && deviceProp.minor==9999) printf("There is no device supporting CUDA.\n");
          else
            if (deviceCount==1) printf("There is 1 device supporting CUDA\n");
            else  printf("There are %d devices supporting CUDA\n",deviceCount);
        }
      printf("\nDevice %d: \"%s\"\n",dev,deviceProp.name);
      printf("  Major revision number:                         %d\n",deviceProp.major);
      printf("  Minor revision number:                         %d\n",deviceProp.minor);
      printf("  Total amount of global memory:                 %u bytes\n",deviceProp.totalGlobalMem);
#if CUDART_VERSION>=2000
      printf("  Number of multiprocessors:                     %d\n",deviceProp.multiProcessorCount);
      printf("  Number of cores:                               %d\n",8*deviceProp.multiProcessorCount);
#endif
      printf("  Total amount of constant memory:               %u bytes\n",deviceProp.totalConstMem);
      printf("  Total amount of shared memory per block:       %u bytes\n",deviceProp.sharedMemPerBlock);
      printf("  Total number of registers available per block: %d\n",deviceProp.regsPerBlock);
      printf("  Warp size:                                     %d\n",deviceProp.warpSize);
      printf("  Maximum number of threads per block:           %d\n",deviceProp.maxThreadsPerBlock);
      printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
	     deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
      printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
	     deviceProp.maxGridSize[0],deviceProp.maxGridSize[1],deviceProp.maxGridSize[2]);
      printf("  Maximum memory pitch:                          %u bytes\n",deviceProp.memPitch);
      printf("  Texture alignment:                             %u bytes\n",deviceProp.textureAlignment);
      printf("  Clock rate:                                    %.2f GHz\n",deviceProp.clockRate * 1e-6f);
#if CUDART_VERSION >=2000
      printf("  Concurrent copy and execution:                 %s\n",deviceProp.deviceOverlap ? "Yes" : "No");
#endif
    }
  
  return(deviceCount);
}
