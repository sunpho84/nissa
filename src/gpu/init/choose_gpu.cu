#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" void choose_gpu()
{
  //check that we have at least one gpu
  if(find_gpu()==0)
    {
      fprintf(stderr,"Error: no gpu found.\n");
      MPI_Abort(MPI_COMM_WORLD,1);
    }
  
  if(hipSetDevice(0)!=hipSuccess)
    {
      fprintf(stderr,"Could not set active device.\n");
      MPI_Abort(MPI_COMM_WORLD,1);
    }  
  
  int dev_num;
  hipGetDevice(&dev_num);
  printf("Rank %d choose device: %d\n",rank,dev_num);
}
