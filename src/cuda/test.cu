#include <stdio.h>

#include "base/global_variables.h"

void cuda_set_device()
{
  //count device
  int ndevices;
  hipError_t stat=hipGetDeviceCount(&ndevices);
  if(stat!=hipSuccess) printf("%s\n",hipGetErrorString(stat));
  printf("Found: %d devices\n",ndevices);
  
  //try to set a device
  for(int idev=0;idev<ndevices;idev++)
    {
      stat=hipSetDevice(idev); 
      if(stat!=hipSuccess) printf("%s\n",hipGetErrorString(stat));
      if(stat==hipErrorInvalidDevice) 
	{ 
	  perror("hipSetDevice returned  hipErrorInvalidDevice"); 
	}
      int device;
      hipGetDevice(&device); 
      printf("hipGetDevice()=%d\n",device); 
    }
}

void cuda_test()
{
  cuda_set_device();
}
