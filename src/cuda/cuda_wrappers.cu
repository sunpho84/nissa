#include "base/debug.hpp"

namespace cuda
{
  //crash translating cuda error
  void crash_on_unsuccess(hipError_t ret)
  {if(ret!=hipSuccess) nissa::crash("%s",hipGetErrorString(ret));}
  
  //return the number of devices
  int get_device_count()
  {
    int ndevices;
    crash_on_unsuccess(hipGetDeviceCount(&ndevices));
    
    return ndevices;
  }
  
  //return used devices
  int get_device()
  {
    int idev;
    crash_on_unsuccess(hipGetDevice(&idev));
    
    return idev;
  }
  
  //set the device to be used
  void set_device(int asked_dev)
  {crash_on_unsuccess(hipSetDevice(asked_dev));}
  
  //copy a float to the symbols
  void memcpy_to_symbol(float dev_float,float host_float)
  {crash_on_unsuccess(hipMemcpyToSymbol(HIP_SYMBOL(dev_float),&host_float,sizeof(float),0,hipMemcpyHostToDevice));}
}
